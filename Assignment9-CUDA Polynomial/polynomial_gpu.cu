
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>

__global__ void polynomial_expansion (float* poly, int degree,
			   int n, float* array) {
  //TODO: Write code to use the GPU here!
  //code should write the output back to array
  
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < n){
    float out = 0.;
    float xtothepowerof = 1.;
    for (int i=0; i<=degree; ++i) {
      out += xtothepowerof*poly[i];
      xtothepowerof *= array[index];
    }
    array[index] = out;

  }

}

int main (int argc, char* argv[]) {
  //TODO: add usage
  
  if (argc < 3) {
     std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
     return -1;
  }

  int n = atoi(argv[1]); //TODO: atoi is an unsafe function
  int degree = atoi(argv[2]);
  int nbiter = 1;

  float* array = new float[n];
  float* poly = new float[degree+1];
  float* d_array;
  float* d_poly;
  for (int i=0; i<n; ++i)
    array[i] = 1.;

  for (int i=0; i<degree+1; ++i)
    poly[i] = 1.;

    long asize=n*sizeof(float);
    long psize=(degree+1)*sizeof(float);


  std::chrono::time_point<std::chrono::system_clock> begin, end;
  begin = std::chrono::system_clock::now();

  hipMallocManaged(&d_array, asize);
  hipMallocManaged(&d_poly, psize);

  hipMemcpy(d_array,array,asize,hipMemcpyHostToDevice);
  hipMemcpy(d_poly,poly,psize,hipMemcpyHostToDevice);

  int blocks=n/256;
  if(n%256>0){
    blocks = n/256+1;
  }

 
  for (int iter = 0; iter<nbiter; ++iter)
    polynomial_expansion<<<blocks,256>>> (d_poly, degree, n, d_array);

  hipMemcpy(array,d_array,asize,hipMemcpyDeviceToHost);
  hipFree(d_array);
  hipFree(d_poly);

  end = std::chrono::system_clock::now();
  std::chrono::duration<double> totaltime = (end-begin)/nbiter;

  {
    bool correct = true;
    int ind;
    for (int i=0; i< n; ++i) {
      if (fabs(array[i]-(degree+1))>0.01) {
        correct = false;
	ind = i;
      }
    }
    if (!correct)
      std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
  }
  

  std::cerr<<array[0]<<std::endl;
  std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

  delete[] array;
  delete[] poly;

  return 0;
}
